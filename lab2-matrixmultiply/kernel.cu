/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    __shared__ float A_tile[TILE_SIZE][TILE_SIZE];
    __shared__ float B_tile[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float Cvalue = 0;

    // Loop over tiles in A and B
    for (int p = 0; p < (k-1)/TILE_SIZE+1; p++)
    {
        // Load tile into shared memory
        if (row < m && p*TILE_SIZE+tx < k)
            A_tile[ty][tx] = A[row*k + p*TILE_SIZE+tx];
        else
            A_tile[ty][tx] = 0.0;
        if (p*TILE_SIZE+ty < k && col < n)
            B_tile[ty][tx] = B[(p*TILE_SIZE+ty)*n + col];
        else
            B_tile[ty][tx] = 0.0;
        __syncthreads();
        
        if (row < m && col < n)
        {
            for (int i = 0; i < TILE_SIZE; i++)
                Cvalue += A_tile[ty][i] * B_tile[i][tx];
        }
        __syncthreads();
    }
    if (row < m && col < n)
        C[row*n + col] = Cvalue;
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
    dim3 dim_grid, dim_block;

    dim_block.x = dim_block.y = BLOCK_SIZE; dim_block.z = 1;
    dim_grid.x = (n - 1) / BLOCK_SIZE + 1;
    dim_grid.y = (m - 1) / BLOCK_SIZE + 1;
    dim_grid.z = 1;

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<dim_grid, dim_block>>>(m, n, k, A, B, C);
}



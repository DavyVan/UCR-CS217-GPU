#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.cu"

const unsigned int numStream = 3;

int main (int argc, char *argv[])
{
    //set standard seed
    srand(217);

    Timer timer;
    hipError_t cuda_ret;

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d[numStream], *B_d[numStream], *C_d[numStream];
    size_t A_sz, B_sz, C_sz;
    unsigned VecSize;

    // Initialize streams
    hipStream_t streams[numStream];
    for (int i = 0; i < numStream; i++)
        hipStreamCreate(&streams[i]);
   
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        VecSize = 1000000;
    } else if (argc == 2) {
        VecSize = atoi(argv[1]);   
    } else {
        printf("\nOh no!\nUsage: ./vecAdd <Size>");
        exit(0);
    }

    A_sz = VecSize;
    B_sz = VecSize;
    C_sz = VecSize;
    const int segmentLen = VecSize / numStream;

    // A_h = (float*) malloc( sizeof(float)*A_sz );
    hipHostAlloc((void**)&A_h, A_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < A_sz; i++) { A_h[i] = (rand()%100)/100.00; }

    // B_h = (float*) malloc( sizeof(float)*B_sz );
    hipHostAlloc((void**)&B_h, B_sz*sizeof(float), hipHostMallocDefault);
    for (unsigned int i=0; i < B_sz; i++) { B_h[i] = (rand()%100)/100.00; }

    // C_h = (float*) malloc( sizeof(float)*C_sz );
    hipHostAlloc((void**)&C_h, C_sz*sizeof(float), hipHostMallocDefault);

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    size Of vector: %u x %u\n  ", VecSize);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    // hipMalloc((float**) &A_d, sizeof(float) * VecSize);
    // hipMalloc((float**) &B_d, sizeof(float) * VecSize);
    // hipMalloc((float**) &C_d, sizeof(float) * VecSize);
    for (int i = 0; i < numStream; i++)
    {
        if (i != numStream-1)
        {
            hipMalloc((float**) &A_d[i], sizeof(float) * segmentLen);
            hipMalloc((float**) &B_d[i], sizeof(float) * segmentLen);
            hipMalloc((float**) &C_d[i], sizeof(float) * segmentLen);
        }
        else    // remainder
        {
            hipMalloc((float**) &A_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
            hipMalloc((float**) &B_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
            hipMalloc((float**) &C_d[i], sizeof(float) * (segmentLen + VecSize % numStream));
        }
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device...\n"); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    // hipMemcpy(A_d, A_h, sizeof(float) * VecSize, hipMemcpyHostToDevice);
    // hipMemcpy(B_d, B_h, sizeof(float) * VecSize, hipMemcpyHostToDevice);
    
    for (int i = 0; i < numStream; i++)
    {
        if (i != numStream-1)
        {
            hipMemcpyAsync(A_d[i], A_h + i*segmentLen, sizeof(float)*segmentLen, hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(B_d[i], B_h + i*segmentLen, sizeof(float)*segmentLen, hipMemcpyHostToDevice, streams[i]);
        }
        else
        {
            hipMemcpyAsync(A_d[i], A_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(B_d[i], B_h + i*segmentLen, sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyHostToDevice, streams[i]);
        }
    }

    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel  ---------------------------
    printf("Launching kernel...\n"); fflush(stdout);
    // startTime(&timer);

    // basicVecAdd(A_d, B_d, C_d, VecSize); //In kernel.cu
    for (int i = 0; i < numStream; i++)
    {
        if (i != numStream-1)
        {
            basicVecAdd(A_d[i], B_d[i], C_d[i], segmentLen, streams[i]);
        }
        else
        {
            basicVecAdd(A_d[i], B_d[i], C_d[i], segmentLen + VecSize % numStream, streams[i]);
        }
    }
        
    // cuda_ret = hipDeviceSynchronize();
	// if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    // startTime(&timer);

    //INSERT CODE HERE
    // hipMemcpy(C_h, C_d, sizeof(float) * VecSize, hipMemcpyDeviceToHost);
    for (int i = 0; i < numStream; i++)
    {
        if (i != numStream-1)
        {
            hipMemcpyAsync(C_h + i*segmentLen, C_d[i], sizeof(float)*segmentLen, hipMemcpyDeviceToHost, streams[i]);
        }
        else
        {
            hipMemcpyAsync(C_h + i*segmentLen, C_d[i], sizeof(float)*(segmentLen + VecSize % numStream), hipMemcpyDeviceToHost, streams[i]);
        }
    }

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, VecSize);


    // Free memory ------------------------------------------------------------

    // free(A_h);
    // free(B_h);
    // free(C_h);
    hipHostFree(A_h);
    hipHostFree(B_h);
    hipHostFree(C_h);

    //INSERT CODE HERE
    // hipFree(A_d);
    // hipFree(B_d);
    // hipFree(C_d);
    for (int i = 0; i < numStream; i++)
    {
        hipFree(A_d[i]);
        hipFree(B_d[i]);
        hipFree(C_d[i]);
        hipStreamDestroy(streams[i]);
    }
    return 0;

}

/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

const unsigned int BLOCK_SIZE = 512; 

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A + B
     *   where A is a (1 * n) vector
     *   where B is a (1 * n) vector
     *   where C is a (1 * n) vector
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}


void basicVecAdd( float *A,  float *B, float *C, int n, hipStream_t stream)
{

    // Initialize thread block and kernel grid dimensions ---------------------

    //INSERT CODE HERE
    dim3 DimGrid((n-1)/BLOCK_SIZE + 1, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    VecAdd <<<DimGrid, DimBlock, 0, stream>>> (n, A, B, C);
}

